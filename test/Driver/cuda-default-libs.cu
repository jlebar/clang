
#include <hip/hip_runtime.h>
// Checks that we add -L /path/to/cuda/lib{,64} and -lcudart_static -ldl -lrt
// -lpthread as appropriate when compiling CUDA code.

// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang -### -v --target=i386-unknown-linux \
// RUN:   -lfoo --sysroot=%S/Inputs/CUDA %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix CUDA -check-prefix CUDA-x86_32
//
// RUN: %clang -### -v --target=x86_64-unknown-linux \
// RUN:   -lfoo --sysroot=%S/Inputs/CUDA %s 2>&1 \
// RUN:   | FileCheck %s -check-prefix CUDA -check-prefix CUDA-x86_64
//
// # Our new flags should come after user-specified flags.
// CUDA: -lfoo
// CUDA-x86_32: "-L" "{{.*}}/Inputs/CUDA/usr/local/cuda/lib"
// CUDA-x86_64: "-L" "{{.*}}/Inputs/CUDA/usr/local/cuda/lib64"
// CUDA-x86_32-NOT: "-L" "{{.*}}/Inputs/CUDA/usr/local/cuda/lib64"
// CUDA-x86_64-NOT: "-L" "{{.*}}/Inputs/CUDA/usr/local/cuda/lib"
// CUDA-DAG: "-lcudart_static"
// CUDA-DAG: "-ldl"
// CUDA-DAG: "-lrt"
// CUDA-DAG: "-lpthread"

// If we can't find CUDA, don't include it in our library search path, and
// don't include any additional libraries via -l.
// RUN: %clang -### -v --target=i386-unknown-linux \
// RUN:   --sysroot=%S/Inputs/no-cuda-there %s 2>&1 | FileCheck %s -check-prefix NOCUDA
//
// Also don't add anything if we pass -nocudalib.
// RUN: %clang -### -v --target=x86_64-unknown-linux -nocudalib \
// RUN:   --sysroot=%S/Inputs/CUDA %s 2>&1 | FileCheck %s -check-prefix NOCUDA
//
// NOCUDA-NOT: "-L" "{{.*}}/no-cuda-there/{{.*}}"
// NOCUDA-NOT: "-lcudart_static"
// NOCUDA-NOT: "-ldl"
// NOCUDA-NOT: "-lrt"
// NOCUDA-NOT: "-lpthread"
// NOCUDA-NOT: "-L" "{{.*}}/Inputs/CUDA/usr/local/cuda/lib"
// NOCUDA-NOT: "-L" "{{.*}}/Inputs/CUDA/usr/local/cuda/lib64"
